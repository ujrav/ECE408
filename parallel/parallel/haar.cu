#include "hip/hip_runtime.h"
// MP Scan
// Given a list (lst) of length n
// Output its prefix sum = {lst[0], lst[0] + lst[1], lst[0] + lst[1] + ... +
// lst[n-1]}

#include "haar.cuh"


using namespace std;

__device__ __constant__ stage_t deviceStages[FEATURENUM];
__device__ __constant__ stageMeta_t deviceStagesMeta[STAGENUM];


//-------------------------------------------------------------------------
// Naive Haar Cascade Kernel (one window scale, no shared memory)
//-------------------------------------------------------------------------
__global__ void naiveCudaHaarKernel(float* deviceIntegralImage, int width, int height, int winWidth, int winHeight, float scale, int step, rectBig_t* deviceResults,int* deviceResultsNum)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int originX;
	int originY;
	int stageIndex;

	originX = (blockIdx.x*BLOCK_SIZE + tx)*step;
	originY = (blockIdx.y*BLOCK_SIZE + ty)*step;

	// for each stage in stagesMeta
	feature_t *feature;
	float third = 0;
	if (originX <= (width - winWidth) && originY <= (height - winHeight)) {
		for (int sIdx = 0; sIdx < STAGENUM; ++sIdx)
		{

			uint8_t stageSize = deviceStagesMeta[sIdx].size;
			float stageThreshold = deviceStagesMeta[sIdx].threshold;
			float featureSum = 0.0;
			float sum;

			// for each classifier in a stage
			for (int cIdx = 0; cIdx < stageSize; ++cIdx)
			{
				// get feature index and threshold
				stageIndex = deviceStagesMeta[sIdx].start + cIdx;
				float featureThreshold = deviceStages[stageIndex].threshold;
				feature = &(deviceStages[stageIndex].feature);

				// get black rectangle of feature fIdx
				uint8_t rectX = feature->black.x * scale;
				uint8_t rectY = feature->black.y * scale;
				uint8_t rectWidth = feature->black.w * scale;
				uint8_t rectHeight = feature->black.h * scale;
				int8_t rectWeight = feature->black.weight;

				float black = (float)rectWeight * rectSum(deviceIntegralImage, width, height, originX + rectX, originY + rectY, rectWidth, rectHeight);

				// get white rectangle of feature fIdx
				rectX = feature->white.x * scale;
				rectY = feature->white.y * scale;
				rectWidth = feature->white.w * scale;
				rectHeight = feature->white.h * scale;
				rectWeight = feature->white.weight;

				float white = (float)rectWeight * rectSum(deviceIntegralImage, width, height, originX + rectX, originY + rectY, rectWidth, rectHeight);

				third = 0;
				if (feature->third.weight){
					rectX = feature->third.x * scale;
					rectY = feature->third.y * scale;
					rectWidth = feature->third.w * scale;
					rectHeight = feature->third.h * scale;
					rectWeight = feature->third.weight;
					third = (float)rectWeight * rectSum(deviceIntegralImage, width, height, originX + rectX, originY + rectY, rectWidth, rectHeight);
				}

				sum = (black + white + third) / ((float)(winWidth * winHeight));

				if (sum > featureThreshold)
					featureSum += deviceStages[stageIndex].rightWeight;
				else
					featureSum += deviceStages[stageIndex].leftWeight;

			}

			if (featureSum < stageThreshold){
				//Failed
				return;
			}

		}
		//printf("Passed at originX: %d originY: %d\n", originX, originY);
		int old = atomicAdd(deviceResultsNum, 1);
		deviceResults[old].x = originX;
		deviceResults[old].y = originY;
		deviceResults[old].w = winWidth;
		deviceResults[old].h = winHeight;
	}
	return;

}

int CudaHaarCascade(unsigned char* outputImage, const float* integralImage, int width, int height)
{
	float scaleWidth = ((float)width) / 20.0f;
	float scaleHeight = ((float)height) / 20.0f;
	int step;
	float scale;
	hipError_t cudaStatus;
	float *deviceIntegralImage;

	rectBig_t results[100];
	int resultsNum = 0;
	rectBig_t* deviceResults;
	int* deviceResultsNum;

	float scaleStart = scaleHeight < scaleWidth ? scaleHeight : scaleWidth;

	int scaleMaxItt = (int)ceil(log(1 / scaleStart) / log(1.0 / 1.2));

	cout << scaleMaxItt << endl;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	//allocate GPU memory
	cudaStatus = hipMalloc((void**)&deviceIntegralImage, height * width * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc of integralImage failed!");
	}
	cudaStatus = hipMalloc((void**)&deviceResults, 100 * sizeof(rectBig_t));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc of integralImage failed!");
	}
	cudaStatus = hipMalloc((void**)&deviceResultsNum, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc of integralImage failed!");
	}

	// copy data to GPU memory
	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(deviceStagesMeta), stagesMeta, stageNum * sizeof(stageMeta_t), 0, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy of StageMeta failed!");
	}

	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(deviceStages), stagesFlat, featureNum * sizeof(stage_t), 0, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy of StageMeta failed!");
	}

	cudaStatus = hipMemcpy(deviceIntegralImage, integralImage, height * width * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy of integralImage failed!");
	}

	cudaStatus = hipMemcpy(deviceResultsNum, &resultsNum, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy of integralImage failed!");
	}

	for (int sIdx = 0; sIdx < scaleMaxItt; ++sIdx)
	{
		scale = scaleStart*(float)powf(1.0f / 1.2f, (float)(sIdx));
		//cout << "Scale: " << scale << endl;

		step = (int)scale > 2 ? (int)scale : 2;

		int winWidth = (int)(20 * scale);
		int winHeight = (int)(20 * scale);

		dim3 DimGridSimple((width - winWidth) / (step * 32) + 1, (height - winHeight) / (step * 32) + 1, 1);
		dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

		//printf("naiveCudaHaarKernel with dimensions %d x %d x %d launching\n", DimGridSimple.x, DimGridSimple.y, DimGridSimple.z);

		naiveCudaHaarKernel << <DimGridSimple, DimBlock >> >(deviceIntegralImage, width, height, winWidth, winHeight, scale, step, deviceResults, deviceResultsNum);
	}
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "naiveCudaHaarKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching naiveCudaHaarKernel!\n", cudaStatus);
	}

	cudaStatus = hipMemcpy(&resultsNum, deviceResultsNum, sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy of integralImage failed!");
	}

	cudaStatus = hipMemcpy(results, deviceResults, 100 * sizeof(rectBig_t), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy of integralImage failed!");
	}

	hipFree(deviceResults);
	hipFree(deviceResultsNum);
	hipFree(deviceIntegralImage);

	for (int n = 0; n < resultsNum; n++){
		int x = results[n].x;
		int y = results[n].y;
		int winWidth = results[n].w;
		int winHeight = results[n].h;
		for (int i = 0; i < winWidth; i++){
			outputImage[3 * (x + i + (y)*width) + 1] = 255;
			outputImage[3 * (x + i + (y + winHeight - 1)*width) + 1] = 255;
		}
		for (int j = 0; j < winHeight; j++){
			outputImage[3 * (x + (y + j)*width) + 1] = 255;
			outputImage[3 * (x + winWidth - 1 + (y + j)*width) + 1] = 255;
		}
	}

	return 0;
}


__device__ __host__ float rectSum(const float* integralImage, int imWidth, int imHeight, int x, int y, int w, int h){
	float a, b, c, d;

	if (x - 1 < 0 || y - 1 < 0){
		a = 0;
	}
	else{
		a = integralImage[(x - 1) + (y - 1) * imWidth];
	}

	if (x - 1 + w < 0 || y - 1 < 0){
		b = 0;
	}
	else{
		b = integralImage[(x - 1 + w) + (y - 1)*imWidth];
	}

	if (x - 1 < 0 || y - 1 + h < 0){
		c = 0;
	}
	else{
		c = integralImage[(x - 1) + (y - 1 + h) * imWidth];
	}

	if (x - 1 + w < 0 || y - 1 + h < 0){
		d = 0;
	}
	else{
		d = integralImage[(x - 1 + w) + (y - 1 + h) * imWidth];
	}

	return (float)(d - c - b + a);
}
